#include <fstream>
#include <filesystem>
#include <chrono>
#include <random>
#include <map>

#include <hip/hip_runtime.h>
#include <>

#include "classify_images.h"
#include "utils.h"

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char* file, int line, bool abort = true)
{
	if (code != hipSuccess)
	{
		fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
		if (abort) exit(code);
	}
}

using namespace cv;
using namespace std;
using namespace chrono;


string kNearestNeighbors(float*& x, vector<float*>& features, vector<string>& labels, int n = 100, int K = 32, int k = 10)
{

	vector<pair<float, string>> dists;

	for (int i = 0; i < features.size(); i += 1) {
		dists.push_back(make_pair(distEuc(x, features[i], K), labels[i]));
	}

	auto comp = [](pair<float, string> p1, pair<float, string> p2) {
		return p1.first < p2.first;
	};

	sort(dists.begin(), dists.end(), comp);

	map<string, int> cnt;

	for (int i = 0; i < k; i += 1) {
		string s = dists[i].second;
		if (cnt.find(s) == cnt.end()) {
			cnt[s] = 1;
		}
		else {
			cnt[s] += 1;
		}
	}

	int maxCnt = 0;
	string label;
	for (auto l : cnt) {
		if (l.second > maxCnt) {
			maxCnt = l.second;
			label = l.first;
		}
	}

	return label;
	 
}


void classifyImagesSeq()
{
	int numFilters = 32;

	vector<string> paths;
	vector<string> labels;
	vector<Mat> images;

	vector<float*> trainFeatures;
	vector<float*> testFeatures;
	vector<string> trainLabels;
	vector<string> testLabels;
	int numCenters;

	loadImages(paths, labels, images);
	loadFeatures(labels, trainLabels, testLabels, trainFeatures, testFeatures, numCenters, 0.2);

	auto start = duration_cast<milliseconds>(system_clock::now().time_since_epoch()).count();

	vector<string> predLabels;

	int correct = 0, total = 0;

#pragma omp parallel for
	for (int i = 0; i < testFeatures.size(); i += 1) {
		string label = kNearestNeighbors(testFeatures[i], trainFeatures, trainLabels, numCenters, numFilters, 25);
		predLabels.push_back(label);
		if (label.compare(testLabels[i]) == 0) {
#pragma omp atomic
			correct += 1;
		}
#pragma omp atomic
		total += 1;
	}

	cout << correct * 1.f / total << endl;

	auto end = duration_cast<milliseconds>(system_clock::now().time_since_epoch()).count();
	cout << "create features computation time (par): " << (end - start) / 1000.0 << endl;

}
